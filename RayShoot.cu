#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hiprand.h>
#include <cstdio>
#include <time.h>
#include <ctime>
#include <optixu/optixu_math_namespace.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "errorchecking.cu"
//
//#define PATCH_NUM 512
//#define SAMPLES 512

struct PatchData {
	optix::float3 a;
	optix::float3 b;
	optix::float3 c;
	optix::float3 norm;
	int id;

};

extern int* main_test(PatchData *patches, int PATCH_NUM, int SAMPLES);
struct Ray {
	optix::float3 orig;	// ray origin
	optix::float3 dir;		// ray direction	
	__device__ Ray(optix::float3 o_, optix::float3 d_) : orig(o_), dir(d_) {}
};

__device__ float RayTriangleIntersection(const Ray &r,
	const optix::float3 &v0, const optix::float3 &edge1, const optix::float3 &edge2) {

	optix::float3 tvec = r.orig - v0;
	optix::float3 pvec = optix::cross(r.dir, edge2);
	float  det = optix::dot(edge1, pvec);

	det = __fdividef(1.0f, det);  // CUDA intrinsic function 

	float u = optix::dot(tvec, pvec) * det;

	if (u < 0.0f || u > 1.0f)
		return -1.0f;

	optix::float3 qvec = optix::cross(tvec, edge1);

	float v = optix::dot(r.dir, qvec) * det;

	if (v < 0.0f || (u + v) > 1.0f)
		return -1.0f;

	return optix::dot(edge2, qvec) * det;
}

//generates random kernel
__global__ void rand_kernel(hiprandState *state, int seed) {
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	hiprand_init(seed, idx, 0, &state[0]);
}

__device__ void intersectAllTriangles(Ray& r, const int number_of_triangles, PatchData *faces, int &face_id) {

	float max_dist = 10000.0f;
	int max_face = -1;
	for (int i = 0; i < number_of_triangles; i++)
	{
		optix::float3 v0 = faces[i].a; 
		optix::float3 edge1 = (faces[i].b - faces[i].a); 
		optix::float3 edge2 = (faces[i].c - faces[i].a); ;

														 // intersect ray with reconstructed triangle	
		float dist = RayTriangleIntersection(r,v0, edge1, edge2);

		// keep track of closest distance and closest triangle
		// if ray/tri intersection finds an intersection point that is closer than closest intersection found so far
		if (dist < max_dist && dist > 0.001)
		{
			max_dist = dist;
			max_face = i;
		}
	}
	face_id = max_face;
	//printf("face_id %d \n", face_id);
}

__global__ void initialise_curand_on_kernels(hiprandState * state, unsigned long seed)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

__device__ float generate(hiprandState* globalState, int ind)
{
	//copy state to local mem
	hiprandState localState = globalState[ind];
	//apply uniform distribution with calculated random
	float rndval = hiprand_uniform(&localState);
	//update state
	globalState[ind] = localState;
	//return value
	return rndval;
}

__global__ void set_random_number_from_kernels(float* _ptr, hiprandState* globalState, const unsigned int _points)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	//only call gen on the kernels we have inited
	//(one per device container element)
	if (idx < _points)
	{
		float x = generate(globalState, idx);
		//printf("%f \n", x);
		_ptr[idx] = x;
	}
}

/*
uses random kernel to calculate ray direction
n : number of rays to generate
num : number of faces
faces[] : array containing all the faces struct
*result : pointer to 2d array of Face -> Array of Directions
*/
__global__ void generate_ray_dir(hiprandState*globalState, PatchData *faces, int num, int *hit) {

	for (int h = 0; h < 32; h++) {
		int idx = threadIdx.x + blockDim.x*blockIdx.x + h;
		//int sad = //threadIdx.x + h;
		int i = blockIdx.x;

		float sin_theta = sqrt(generate(globalState, idx));
		float cos_theta = sqrt(1 - sin_theta * sin_theta);
		float psi = 2 * 3.14159265359 * generate(globalState, idx);
		float a1 = sin_theta * cos(psi);
		float b1 = sin_theta * sin(psi);
		float c1 = cos_theta;


		optix::float3 v1 = a1 * (faces[i].b - faces[i].a);
		optix::float3 v2 = b1 * (faces[i].c - faces[i].a);
		optix::float3 v3 = c1 * faces[i].norm;

		float r2 = generate(globalState,idx);
		float r1 = generate(globalState, idx);

		optix::float3 pt = (1.0 - sqrt(r1))*faces[i].a + (sqrt(r1) * (1.0 - r2))*faces[i].b + (r2 * sqrt(r1)*faces[i].c);
		optix::float3 direction = v1 + v2 + v3;

		Ray ray = Ray(pt, direction);
		int face = 0;
		intersectAllTriangles(ray, num, faces, face);
		//printf("face %d for core %d r2 %f\n", face, blockIdx.x, r2);
		if (threadIdx.x == 0 && blockIdx.x == 0) {
		
			printf("%d %d %d %d\n", blockIdx.x, threadIdx.x, h, face);
		}
		if (face == -1 && threadIdx.x == 0 && blockIdx.x==0) {
			printf("missed %d %d %d\n", blockIdx.x, threadIdx.x, h);
			h -= 1;
		}
		if (blockIdx.x == 100) {
			printf("random # %f", r2);
		}
		else {
			hit[idx] = face;
		}

	}

}

int* main_test(PatchData *patches, int PATCH_NUM, int SAMPLES) {
	PatchData *g_patch_arr = (PatchData*)malloc(PATCH_NUM * sizeof(PatchData));
	//optix::float3 *g_dir_arr = (optix::float3*)malloc(SAMPLES*PATCH_NUM * sizeof(optix::float3)), *g_pt_arr = (optix::float3*)malloc(SAMPLES*PATCH_NUM * sizeof(optix::float3));
	hipMalloc((void**)&g_patch_arr, PATCH_NUM * sizeof(PatchData));
	CudaCheckError();

	int* g_hit = (int*)malloc(SAMPLES*PATCH_NUM * sizeof(int));
	hipMalloc((void**)&g_hit, SAMPLES*PATCH_NUM * sizeof(int));
	CudaCheckError();

	int *c_hit = (int*)malloc(SAMPLES*PATCH_NUM * sizeof(int));
	hipMemcpy(g_patch_arr, patches, PATCH_NUM * sizeof(PatchData), hipMemcpyHostToDevice);
	CudaCheckError();
	std::clock_t start;
	float duration;


	hiprandState* deviceStates;
	printf("size of hiprand %d\n", sizeof(hiprandState));
	hipMalloc((void**)&deviceStates, PATCH_NUM * sizeof(hiprandState));
	CudaCheckError();

	initialise_curand_on_kernels << <PATCH_NUM, SAMPLES / 16 >> > (deviceStates, unsigned(time(NULL)));
	hipDeviceSynchronize();

	CudaCheckError();

	start = std::clock();

	generate_ray_dir << <PATCH_NUM, SAMPLES/16 >> > (deviceStates, g_patch_arr, PATCH_NUM, g_hit);
	hipDeviceSynchronize();
	CudaCheckError();

	hipMemcpy(c_hit, g_hit, SAMPLES*PATCH_NUM * sizeof(int), hipMemcpyDeviceToHost);
	CudaCheckError();

	duration = (std::clock() - start) / (float)CLOCKS_PER_SEC;
	printf("%f\n", duration);

	hipFree(g_patch_arr);
	CudaCheckError();
	/*hipFree(g_dir_arr);
	CudaCheckError();*/
	hipFree(deviceStates);
	CudaCheckError();
	hipFree(g_hit);
	CudaCheckError();
	return c_hit;
}


