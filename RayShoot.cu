#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hiprand.h>
#include <cstdio>
#include <time.h>
#include <ctime>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "errorchecking.cu"

#define PATCH_NUM 40
#define SAMPLES 1024

struct PerRayData_pathtrace
{
	optix::float3 origin;
	optix::float3 direction;
	optix::float3 result;
	float importance;
	int depth;
};

struct PatchData {
	optix::float3 a;
	optix::float3 b;
	optix::float3 c;
	optix::float3 norm;
	int id;
};

//generates random kernel
__global__ void rand_kernel(hiprandState *state, int seed) {
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

/*
uses random kernel to calculate ray direction 
n : number of rays to generate
num : number of faces
faces[] : array containing all the faces struct
*result : pointer to 2d array of Face -> Array of Directions 
*/
__global__ void generate_ray_dir(hiprandState *rand1, hiprandState *rand2, int n, PatchData *faces, int num, optix::float3 *dir, optix::float3 *pts) {

	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	int i = blockIdx.x;
	int count = 0;
/*
	for (int i = 0; i < num; i++) {
		while (count < n) {*/
	float sin_theta = sqrt(hiprand_uniform(rand1 + idx));
	float cos_theta = sqrt(1 - sin_theta * sin_theta);
	float psi = 2 * 3.14159265359 * hiprand_uniform((rand1  + idx));
	float a1 = sin_theta * cos(psi);
	float b1 = sin_theta * sin(psi);
	float c1 = cos_theta;

			
	optix::float3 v1 = a1 * (faces[i].b - faces[i].a);
	optix::float3 v2 = b1 * (faces[i].c - faces[i].a);
	optix::float3 v3 = c1 * faces[i].norm;
			
	float r2 = hiprand_uniform((rand2 + idx));
	float r1 = hiprand_uniform((rand2  + idx));

	optix::float3 pt = (1.0 - sqrt(r1))*faces[i].a + (sqrt(r1) * (1.0 - r2))*faces[i].b + (r2 * sqrt(r1)*faces[i].c); 
	optix::float3 direction = v1 + v2 + v3;
	dir[idx] = direction; //optix::make_float3(0.5f, 0.5f, 2.5* idx); //
	//int pos = i * n + count;
	//optix::float3 pt2 = optix::make_float3(0.5f, 0.5f, idx);
	pts[idx] = pt;//
	//		count++;
	//	}
	//}
	//count = 0;
}

int main() {
	hiprandState *d_state;
	hipMalloc((void**)&d_state, sizeof(hiprandState));
	CudaCheckError();
	hiprandState *d_state1;
	hipMalloc((void**)&d_state1, sizeof(hiprandState));
	CudaCheckError();
	srand(time(NULL));
	rand_kernel <<<1, 1 >>>(d_state, rand());
	CudaCheckError();
	srand(time(NULL));
	rand_kernel <<<1, 1 >>>(d_state1, rand());
	CudaCheckError();

	//host data structures
	PatchData test;
	PatchData *patches = (PatchData*)malloc(PATCH_NUM * sizeof(PatchData));
	int p = 0;
	for (p = 0; p < PATCH_NUM; p++) {
		PatchData *t = (PatchData*)malloc(sizeof(PatchData));
		t->a = optix::make_float3(1.0f, 1.0f, 1.0f);
		t->b = optix::make_float3(1.0f, 0.0f, 1.0f);
		t->c = optix::make_float3(1.0f, 1.0f, 0.0f);
		t->norm = optix::make_float3(0.0f, -1.0f, 0.0f);
		patches[p] = *t;	
	}

	optix::float3 *c_dir_arr = (optix::float3*)malloc(SAMPLES *PATCH_NUM * sizeof(optix::float3));
	optix::float3 *c_pt_arr = (optix::float3*)malloc(SAMPLES*PATCH_NUM * sizeof(optix::float3));


	//device data structures
	PatchData *g_patch_arr= (PatchData*)malloc(PATCH_NUM * sizeof(PatchData));
	optix::float3 *g_dir_arr= (optix::float3*)malloc(SAMPLES*PATCH_NUM * sizeof(optix::float3)) , *g_pt_arr=(optix::float3*)malloc(SAMPLES*PATCH_NUM * sizeof(optix::float3));
	hipMalloc((void**)&g_patch_arr, PATCH_NUM*sizeof(PatchData));
	CudaCheckError();

	hipMalloc((void**)&g_dir_arr, SAMPLES * PATCH_NUM * sizeof(optix::float3));
	CudaCheckError();
	hipMalloc((void**)&g_pt_arr, SAMPLES*PATCH_NUM * sizeof(optix::float3));
	CudaCheckError();

	hipMemcpy( g_patch_arr, patches, PATCH_NUM * sizeof(PatchData), hipMemcpyHostToDevice);
	CudaCheckError();

	//dim3 grid(1, 1, 1);
	//dim3 threads((PATCH_NUM + 255) / 256, 1, 1);
	//int threads = (PATCH_NUM+63) / 64;
	std::clock_t start;
	float duration;

	start = std::clock();

	generate_ray_dir <<<PATCH_NUM, 1024>>> (d_state, d_state1, SAMPLES, g_patch_arr, PATCH_NUM, g_dir_arr, g_pt_arr);
	CudaCheckError();
	hipMemcpy(c_dir_arr, g_dir_arr, SAMPLES*PATCH_NUM * sizeof(optix::float3), hipMemcpyDeviceToHost);
	hipMemcpy(c_pt_arr, g_pt_arr, SAMPLES*PATCH_NUM * sizeof(optix::float3), hipMemcpyDeviceToHost);
	duration = (std::clock() - start) / (float)CLOCKS_PER_SEC;
	printf("%f\n", duration);
	//for (int i = 0; i < PATCH_NUM; i++) {
	//	for (int j = 0; j < SAMPLES; j++) {
	float f = c_pt_arr[(PATCH_NUM - 1)*SAMPLES + SAMPLES-1].z;
	float f2 = c_dir_arr[(PATCH_NUM - 1)*SAMPLES + SAMPLES - 1].z;
	printf("%f , %f \n", f, f2);
	/*	}
		printf("\n");
	}*/
	printf("Done");
	
	free(patches);
	free(c_dir_arr);
	free(c_pt_arr);
	return 0;
}



