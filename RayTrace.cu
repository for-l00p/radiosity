#include "hip/hip_runtime.h"


#include <optixu/optixu_math_namespace.h>
using namespace optix;

struct PerRayData_pathtrace
{
	float3 origin;
	float3 direction;
	int depth;
};


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(int , hit_face, rtIntersection);

RT_PROGRAM void setRequiredValues() {



}

RT_PROGRAM void rayTrace() {

	for (;;)
	{
		Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, 0.0001f, RT_DEFAULT_MAX);
		rtTrace(top_object, ray, prd);

		if (prd.done)
		{
			// We have hit the background or a luminaire
			prd.result += prd.radiance * prd.attenuation;
			break;
		}

		// Russian roulette termination 
		if (prd.depth >= rr_begin_depth)
		{
			float pcont = fmaxf(prd.attenuation);
			if (rnd(prd.seed) >= pcont)
				break;
			prd.attenuation /= pcont;
		}

		prd.depth++;
		prd.result += prd.radiance * prd.attenuation;

		// Update ray data for the next path segment
		ray_origin = prd.origin;
		ray_direction = prd.direction;
	}

	result += prd.result;
	seed = prd.seed;
}


}

