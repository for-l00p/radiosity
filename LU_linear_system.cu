#include <stdio.h>

#include "hip/hip_runtime.h" 
#include ""

#include "hipblas.h"

#include "Utilities.cuh"

float *createPermutationMatrix(int *h_PivotArray, int N)
{
	int temp;

	// --- Create permutation matrix
	float *P = (float *)malloc(N * N * sizeof(float));
	for (int i = 0; i<N; i++) {
		P[i] = 0.0f;
		for (int j = 0; j<N; j++)
			if (i == j) P[i * N + j] = 1.0f;
	}

	for (int j = 0; j<N; j++)
		for (int i = 0; i<N - 1; i++) {
			temp = P[i + j * N];
			P[i + j * N] = P[(h_PivotArray[i] - 1) + j * N];
			P[(h_PivotArray[i] - 1) + j * N] = temp;
		}

	return P;
}

/********/
/* MAIN */
/********/
int main() {

	const unsigned int N = 3;

	const unsigned int Nmatrices = 1;

	hipblasHandle_t handle;
	cublasSafeCall(hipblasCreate(&handle));

	/***********************/
	/* SETTING THE PROBLEM */
	/***********************/

	// --- Matrices to be inverted (only one in this example)
	float *h_A = new float[N*N*Nmatrices];

	h_A[0] = 4.f;
	h_A[1] = 3.f;
	h_A[2] = 8.f;
	h_A[3] = 9.f;
	h_A[4] = 5.f;
	h_A[5] = 1.f;
	h_A[6] = 2.f;
	h_A[7] = 7.f;
	h_A[8] = 6.f;

	// --- Known term (only one in this example)
	float *h_B = new float[N];

	h_B[0] = 1.f;
	h_B[1] = 0.5f;
	h_B[2] = 3.;

	// --- Result (only one in this example)
	float *h_X = new float[N];

	// --- Allocate device space for the input matrices 
	float *d_A;	gpuErrchk(hipMalloc((void**)&d_A, N*N*Nmatrices * sizeof(float)));
	float *d_B;	gpuErrchk(hipMalloc((void**)&d_B, N * sizeof(float)));
	float *d_X;	gpuErrchk(hipMalloc((void**)&d_X, N * sizeof(float)));

	// --- Move the relevant matrices from host to device
	gpuErrchk(hipMemcpy(d_A, h_A, N*N*Nmatrices * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice));

	/********************/
	/* LU DECOMPOSITION */
	/********************/

	// --- Creating the array of pointers needed as input/output to the batched getrf
	float **h_inout_pointers = (float **)malloc(Nmatrices * sizeof(float *));
	for (int i = 0; i<Nmatrices; i++) h_inout_pointers[i] = (float *)((char*)d_A + i * ((size_t)N*N) * sizeof(float));

	float **d_inout_pointers;
	gpuErrchk(hipMalloc((void**)&d_inout_pointers, Nmatrices * sizeof(float *)));
	gpuErrchk(hipMemcpy(d_inout_pointers, h_inout_pointers, Nmatrices * sizeof(float *), hipMemcpyHostToDevice));
	free(h_inout_pointers);

	int *d_PivotArray; gpuErrchk(hipMalloc((void**)&d_PivotArray, N*Nmatrices * sizeof(int)));
	int *d_InfoArray;  gpuErrchk(hipMalloc((void**)&d_InfoArray, Nmatrices * sizeof(int)));

	int *h_PivotArray = (int *)malloc(N*Nmatrices * sizeof(int));
	int *h_InfoArray = (int *)malloc(Nmatrices * sizeof(int));

	cublasSafeCall(hipblasSgetrfBatched(handle, N, d_inout_pointers, N, d_PivotArray, d_InfoArray, Nmatrices));
	//cublasSafeCall(hipblasSgetrfBatched(handle, N, d_inout_pointers, N, NULL, d_InfoArray, Nmatrices));

	gpuErrchk(hipMemcpy(h_InfoArray, d_InfoArray, Nmatrices * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < Nmatrices; i++)
		if (h_InfoArray[i] != 0) {
			fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
			hipDeviceReset();
			exit(EXIT_FAILURE);
		}

	gpuErrchk(hipMemcpy(h_A, d_A, N*N * sizeof(float), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_PivotArray, d_PivotArray, N*Nmatrices * sizeof(int), hipMemcpyDeviceToHost));

	// --- The output factored matrix in column-major format
	for (int i = 0; i<N*N; i++) printf("A[%i]=%f\n", i, h_A[i]);

	printf("\n\n");
	// --- The pivot array
	for (int i = 0; i<N; i++) printf("IPIV[%i]=%i\n", i, h_PivotArray[i]);

	/*******************************************/
	/* APPROACH NR.1: THROUGH THE INVERSE OF A */
	/*******************************************/

	// --- Allocate host space for the inverted matrices 
	float *h_C = new float[N*N*Nmatrices];

	// --- Allocate device space for the inverted matrices 
	float *d_C;	gpuErrchk(hipMalloc((void**)&d_C, N*N*Nmatrices * sizeof(float)));

	// --- Creating the array of pointers needed as output to the batched getri
	float **h_out_pointers = (float **)malloc(Nmatrices * sizeof(float *));
	for (int i = 0; i<Nmatrices; i++) h_out_pointers[i] = (float *)((char*)d_C + i * ((size_t)N*N) * sizeof(float));

	float **d_out_pointers;
	gpuErrchk(hipMalloc((void**)&d_out_pointers, Nmatrices * sizeof(float *)));
	gpuErrchk(hipMemcpy(d_out_pointers, h_out_pointers, Nmatrices * sizeof(float *), hipMemcpyHostToDevice));
	free(h_out_pointers);

	cublasSafeCall(hipblasSgetriBatched(handle, N, (const float **)d_inout_pointers, N, d_PivotArray, d_out_pointers, N, d_InfoArray, Nmatrices));

	gpuErrchk(hipMemcpy(h_InfoArray, d_InfoArray, Nmatrices * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < Nmatrices; i++)
		if (h_InfoArray[i] != 0) {
			fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
			hipDeviceReset();
			exit(EXIT_FAILURE);
		}

	gpuErrchk(hipMemcpy(h_C, d_C, N*N * sizeof(float), hipMemcpyDeviceToHost));

	// --- The output inverted matrix in column-major format
	printf("\n\n");
	for (int i = 0; i<N*N; i++) printf("C[%i]=%f\n", i, h_C[i]);

	float alpha1 = 1.f;
	float beta1 = 0.f;

	cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &alpha1, d_C, N, d_B, 1, &beta1, d_X, 1));

	gpuErrchk(hipMemcpy(h_X, d_X, N * sizeof(float), hipMemcpyDeviceToHost));

	// --- The output inverted matrix in column-major format
	printf("\n\n");
	for (int i = 0; i<N; i++) printf("X[%i]=%f\n", i, h_X[i]);

	/*****************************************************************************/
	/* APPROACH NR.2: THROUGH THE INVERSE OF UPPER AND LOWER TRIANGULAR MATRICES */
	/*****************************************************************************/

	float *P = createPermutationMatrix(h_PivotArray, N);

	float *d_P; gpuErrchk(hipMalloc((void**)&d_P, N * N * sizeof(float)));

	printf("\n\n");
	// --- The permutation matrix
	for (int i = 0; i<N; i++)
		for (int j = 0; j<N; j++)
			printf("P[%i, %i]=%f\n", i, j, P[j * N + i]);

	gpuErrchk(hipMemcpy(d_P, P, N * N * sizeof(float), hipMemcpyHostToDevice));

	// --- Now P*A=L*U
	//     Linear system A*x=y => P.'*L*U*x=y => L*U*x=P*y

	cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &alpha1, d_P, N, d_B, 1, &beta1, d_B, 1));

	gpuErrchk(hipMemcpy(h_B, d_B, N * sizeof(float), hipMemcpyDeviceToHost));

	// --- The result of P*y
	printf("\n\n");
	for (int i = 0; i<N; i++) printf("(P*y)[%i]=%f\n", i, h_B[i]);

	// --- 1st phase - solve Ly = b 
	const float alpha = 1.f;

	// --- Function solves the triangulatr linear system with multiple right hand sides, function overrides b as a result 

	// --- Lower triangular part
	cublasSafeCall(hipblasStrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, 1, &alpha, d_A, N, d_B, N));

	// --- Upper triangular part
	cublasSafeCall(hipblasStrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, N, 1, &alpha, d_A, N, d_B, N));

	gpuErrchk(hipMemcpy(h_B, d_B, N * sizeof(float), hipMemcpyDeviceToHost));

	// --- The output inverted matrix in column-major format
	printf("\n\n");
	for (int i = 0; i<N; i++) printf("B[%i]=%f\n", i, h_B[i]);

	return 0;
}