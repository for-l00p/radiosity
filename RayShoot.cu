#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hiprand.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "errorchecking.cu"

#define PATCH_NUM 1
#define SAMPLES 5

struct PerRayData_pathtrace
{
	optix::float3 origin;
	optix::float3 direction;
	optix::float3 result;
	float importance;
	int depth;
};

struct PatchData {
	optix::float3 a;
	optix::float3 b;
	optix::float3 c;
	optix::float3 norm;
	int id;
};

//generates random kernel
__global__ void rand_kernel(hiprandState *state, int seed) {
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

/*
uses random kernel to calculate ray direction 
n : number of rays to generate
num : number of faces
faces[] : array containing all the faces struct
*result : pointer to 2d array of Face -> Array of Directions 
*/
__global__ void generate_ray_dir(hiprandState *rand1, hiprandState *rand2, int n, PatchData *faces, int num, optix::float3 *dir, optix::float3 *pts) {

	int idx = threadIdx.x + blockDim.x*blockIdx.x;

	int count = 0;

	for (int i = 0; i < num; i++) {
		while (count < n) {
			float sin_theta = sqrt(hiprand_uniform(rand1 + idx));
			float cos_theta = sqrt(1 - sin_theta * sin_theta);
			float psi = 2 * 3.14159265359 * hiprand_uniform((rand1  + idx));
			float a1 = sin_theta * cos(psi);
			float b1 = sin_theta * sin(psi);
			float c1 = cos_theta;

			
			optix::float3 v1 = a1 * (faces[i].b - faces[i].a);
			optix::float3 v2 = b1 * (faces[i].c - faces[i].a);
			optix::float3 v3 = c1 * faces[i].norm;
			
			float r2 = hiprand_uniform((rand2 + idx));
			float r1 = hiprand_uniform((rand2  + idx));

			optix::float3 pt = (1.0 - sqrt(r1))*faces[i].a + (sqrt(r1) * (1.0 - r2))*faces[i].b + (r2 * sqrt(r1)*faces[i].c); 
			optix::float3 direction = v1 + v2 + v3;
			dir[i*n + count] = direction; //optix::make_float3(0.5f, 0.5f, 2.5* count); //
			int pos = i * n + count;
			optix::float3 pt2 = pt;//optix::make_float3(0.5f, 0.5f, count);
			pts[pos] = pt2;//
			count++;
		}
	}
	count = 0;
}

int main() {
	hiprandState *d_state;
	hipMalloc((void**)&d_state, sizeof(hiprandState));
	CudaCheckError();
	hiprandState *d_state1;
	hipMalloc((void**)&d_state1, sizeof(hiprandState));
	CudaCheckError();
	srand(time(NULL));
	rand_kernel <<<1, 1 >>>(d_state, rand());
	CudaCheckError();
	srand(time(NULL));
	rand_kernel <<<1, 1 >>>(d_state1, rand());
	CudaCheckError();

	//host data structures
	PatchData test;
	PatchData *patches = (PatchData*)malloc(PATCH_NUM * sizeof(PatchData));
	PatchData *t = (PatchData*)malloc(sizeof(PatchData));
	t->a = optix::make_float3(1.0f, 1.0f, 1.0f);
	t->b = optix::make_float3(1.0f, 0.0f, 1.0f);
	t->c = optix::make_float3(1.0f, 1.0f, 0.0f);
	t->norm = optix::make_float3(0.0f,-1.0f,0.0f);
	patches[0] = *t;

	optix::float3 *c_dir_arr = (optix::float3*)malloc(SAMPLES *PATCH_NUM * sizeof(optix::float3));
	optix::float3 *c_pt_arr = (optix::float3*)malloc(SAMPLES*PATCH_NUM * sizeof(optix::float3));


	//device data structures
	PatchData *g_patch_arr= (PatchData*)malloc(PATCH_NUM * sizeof(PatchData));
	optix::float3 *g_dir_arr= (optix::float3*)malloc(SAMPLES*PATCH_NUM * sizeof(optix::float3)) , *g_pt_arr=(optix::float3*)malloc(SAMPLES*PATCH_NUM * sizeof(optix::float3));
	hipMalloc((void**)&g_patch_arr, PATCH_NUM*sizeof(PatchData));
	CudaCheckError();

	hipMalloc((void**)&g_dir_arr, SAMPLES * PATCH_NUM * sizeof(optix::float3));
	CudaCheckError();
	hipMalloc((void**)&g_pt_arr, SAMPLES*PATCH_NUM * sizeof(optix::float3));
	CudaCheckError();

	hipMemcpy( g_patch_arr, patches, PATCH_NUM * sizeof(PatchData), hipMemcpyHostToDevice);
	CudaCheckError();

	//dim3 grid(1, 1, 1);
	//dim3 threads(num_threads, 1, 1);
	//dim3 threads2(len, 1, 1); // more threads needed fir separate int2 version
	//						  // execute the kernel
	//kernel << < grid, threads >> >((int *)d_data);

	dim3 grid(1, 1, 1);
	dim3 threads((PATCH_NUM + 63) / 64, 1, 1);

	generate_ray_dir <<<grid, threads>>> (d_state, d_state1, SAMPLES, g_patch_arr, PATCH_NUM, g_dir_arr, g_pt_arr);
	CudaCheckError();
	hipMemcpy(c_dir_arr, g_dir_arr, SAMPLES*PATCH_NUM * sizeof(optix::float3), hipMemcpyDeviceToHost);
	hipMemcpy(c_pt_arr, g_pt_arr, SAMPLES*PATCH_NUM * sizeof(optix::float3), hipMemcpyDeviceToHost);
	for (int i = 0; i < PATCH_NUM; i++) {
		for (int j = 0; j < SAMPLES; j++) {
			float f = c_pt_arr[i*SAMPLES + j].z;
			float f2 = c_dir_arr[i*SAMPLES + j].z;
			printf("%f , %f \n", f, f2);
		}
		printf("\n");
	}
	printf("Done");
	
	free(patches);
	free(c_dir_arr);
	free(c_pt_arr);
	return 0;
}



